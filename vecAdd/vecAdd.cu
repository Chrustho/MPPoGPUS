
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float *C, float *A, float *B, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

void vecAdd(float *C, float *A, float *B, int n)
{
    float *d_C, *d_B, *d_A;
    int size = n * sizeof(float);
    int block_size = 32, number_of_blocks = ceil((float)n / block_size);

    hipMalloc((void **)&d_A, size); //<-- Async
    hipMalloc((void **)&d_B, size); //<-- Async
    hipMalloc((void **)&d_C, size); //<-- Async

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    vecAddKernel<<<number_of_blocks, block_size>>>(d_C, d_A, d_B, n); //<-- Async

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C); //<-- Async
}

int main()
{
    int N = 4;
    float h_A[] = {16.0f, 17.0f, 18.0f, 19.0f};
    float h_B[] = {256.0f, 256.0f, 256.0f, 256.0f};
    float *h_C = (float *)malloc(N * sizeof(float));
    vecAdd(h_C, h_A, h_B, N);

    for (size_t i = 0; i < N; i++)
    {
        printf("%f\t", h_C[i]);
    }
    printf("\n");
}